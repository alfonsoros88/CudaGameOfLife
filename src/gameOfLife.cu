#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "gameOfLife.h"

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

__global__ void gameOfLifeIt(unsigned char* d_dst, unsigned char* d_buff, const size_t width, const size_t height) {
    extern __shared__ unsigned char board_sh[];

    size_t glob_x = blockDim.x * blockIdx.x + threadIdx.x;
    size_t glob_y = blockDim.y * blockIdx.y + threadIdx.y;
    size_t glob_idx = glob_y * width + glob_x;

    size_t index = blockDim.x * threadIdx.y + threadIdx.x;

    int share_width = blockDim.x + 2;
    int share_height = blockDim.y + 2;
    int share_size = share_width * share_height;

    bool isActive = (glob_x < width && glob_y < height);
    
    // Copy board to shared memory
    for (int share_idx = index; share_idx < share_size; share_idx += (blockDim.x * blockDim.y)) {

        int x_img = (blockDim.x * blockIdx.x - 1) + (share_idx % share_width);
        int y_img = (blockDim.y * blockIdx.y - 1) + (share_idx / share_width);

        if (x_img < 0) {
            x_img = width - 1;
        } else if (x_img > width - 1) {
            x_img = 0;
        }

        if (y_img < 0) {
            y_img = height - 1;
        } else if (y_img > height - 1) {
            y_img = 0;
        }

        board_sh[share_idx] = d_dst[width * y_img + x_img];
    }

    __syncthreads();

    if (isActive) {

        unsigned char me = board_sh[share_width * (threadIdx.y + 1) + threadIdx.x + 1];
   
        int count = 0;
        for (int i = -1; i < 2; i++) {
            for (int j = -1; j < 2; j++) {
                count += board_sh[share_width * (threadIdx.y + 1 + j) + threadIdx.x + 1 + i];
            }
        } 
        count -= me;

        // Game of life rules
        if (me == 1) {
            if (count < 2) {
                d_buff[glob_idx] = 0;
            }
            else if (count < 4) {
                d_buff[glob_idx] = 1;
            }
            else {
                d_buff[glob_idx] = 0;
            }
        } else {
            if (count == 3) {
                d_buff[glob_idx] = 1;
            }
        }
    }
}

void runGameOfLife(unsigned char* d_dst, unsigned char* d_buff, const size_t width, const size_t height) {
    dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 grid((width + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, (height + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y);

    size_t shared_size = (threads.x + 2) * (threads.y + 2);

    gameOfLifeIt<<<threads, grid, shared_size>>>(d_dst, d_buff, width, height);
    hipMemcpy(d_dst, d_buff, width * height * sizeof(unsigned char), hipMemcpyDeviceToDevice);
}
